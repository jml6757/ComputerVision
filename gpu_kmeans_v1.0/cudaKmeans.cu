#include "hip/hip_runtime.h"
/*
 * File:
 * cudaKmeans.cu
 * version 1.0
 *
 * DESCRIPTION:
 * This is an implementation of k-means clustering in NVIDIA's CUDA.
 * It is designed to interface with MATLAB to speed up MATLAB's computations.
 * 
 * AUTHOR:
 * Nikolaos Sismanis
 *
 * Aristotle University of Thessaloniki
 * Faculty of Engineering
 * Department of Electrical and Computer Engineering
 * Computer Archtecture Lab
 * 
 * DATE:
 * Jan 2011
 *
 * CONTACT INFO:
 * e-mail: nik_sism@hotmail.com nsismani@auth.gr
 */


#include "mex.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include "hip/hip_runtime.h"

#define max_iterations 50
#define BlockSize 512
#define NUMBER_OF_SUBMATRICES 128


typedef struct{
  float *dataset;
  float *members;
  int leading_dim;
  int secondary_dim;
} data_struct;

const char *help="Error using cudaKmeans: 4 arguments\n"
"First (single): dataset, an [mxn] matrix. m must be a multiplier of 128 and n equal or smaller of 512\n"
"Second (single): dataset' ,the transposed dataset an [nxm] matrix\n"
"Third (double): The number of clusters\n"
"Fourth (single): The starting centers (optional)\n";

void errorMessage1(int numarg){

  if(numarg < 3){
    mexErrMsgTxt(help);
  }

}

void errorMessage2(int Objects, int attributes, int numCluster){

  if((Objects % NUMBER_OF_SUBMATRICES) != 0 || attributes > 512 || numCluster >= Objects){
    mexErrMsgTxt(help);
  }
}

void initialize_clusters_rand(data_struct *data_in, data_struct *cluster_in){

  int i, pick = 0;

  int n = cluster_in->leading_dim;
  int m = cluster_in->secondary_dim;
  int Objects = data_in->secondary_dim;
  float *tmp_Centroids = cluster_in->dataset;
  float *tmp_dataset = data_in->dataset;

  srand(time(NULL));
  /*randomly pick initial cluster centers*/
  for(i=0; i<m; i++){
    pick = rand() % Objects;
    tmp_Centroids = cluster_in->dataset + i*n;
    tmp_dataset = data_in->dataset + pick*n;
    memcpy(tmp_Centroids, tmp_dataset, n*sizeof(float));
  }

}

void initialize_clusters(float *buff, data_struct *clusters){

  memcpy(clusters->dataset, buff, clusters->leading_dim*clusters->secondary_dim*sizeof(float));

}

void initialize_device_memory(data_struct *host, data_struct *device){

  hipMemcpy(device->dataset, host->dataset, host->leading_dim*host->secondary_dim*sizeof(float), hipMemcpyHostToDevice);

}


void cleanDevice(data_struct *data){

  hipFree(data->dataset);
  hipFree(data->members);

}

/* for debug only */
void print(data_struct* data2print){

  int i, j = 0;
  int n = data2print->leading_dim;
  int m = data2print->secondary_dim;
  float *tmp_dataset = data2print->dataset;

  
  for(i=0; i<m; i++){
    for(j=0; j<n; j++){
      printf("%f ", tmp_dataset[i*n + j]);
    }
    printf("\n");
  }

  printf("\n");
  
}


__device__ float euclidean_distance_gpu(float *v1, float *v2, int attributes, int numObjects){

  float dist = 0;
  
#pragma unroll 2
  for( int i = 0; i < attributes; i++ )
    {
      float tmp = v2[i*numObjects] - v1[i];
      dist += tmp * tmp;
    }
  return dist;
}


__global__ void CreateClusters(float *dataset, float *centroids, float *index, int *Cond, int numObjects, int numAttributes, int numClusters){

  extern __shared__ float means[];

  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  int element = tid;
  float tmp, minDist = FLT_MAX;

  int tmp_index = -1;


  if(tid<numObjects){
    for(int center=0; center<numClusters; center++){
      tmp = 0;
      
      if(threadIdx.x<numAttributes){
	means[threadIdx.x] = centroids[center*numAttributes + threadIdx.x];
      }
      __syncthreads();
      
      tmp = euclidean_distance_gpu( means, dataset+element ,numAttributes, numObjects);

      //__syncthreads();
      
      if(tmp < minDist){
	minDist = tmp;
	tmp_index = center;
      }
    }
    //__syncthreads();

    if(index[tid] != tmp_index){
      Cond[0] = 1;
    }

    index[tid] = tmp_index;

  }

}


__global__ void submaticesSum_kernel(float *datasetT, float *Index,
float *centroids, float *ClusterSizes, int numObjects, int numAttributes,
int numClusters){

  
  int tmp_index = 0;
  int submatrix_dim = numObjects/gridDim.x;
  float* block_data = datasetT + blockIdx.x*submatrix_dim*numAttributes;
  float* block_clusterIndex = Index + blockIdx.x*submatrix_dim;
  float* tempBlockSum = centroids + blockIdx.x*numClusters*numAttributes;
  float* tempBlockclusterCount = ClusterSizes + blockIdx.x*numClusters; 

  /*Zero Sum and clusterCount*/
#pragma unroll 2
  for(int i=0; i<numClusters; i++){
    tempBlockSum[i*numAttributes + threadIdx.x] = 0;
    if(threadIdx.x==0){
      tempBlockclusterCount[i] = 0;
    }
    __syncthreads();
  }
  
  //__syncthreads();
#pragma unroll 8 
  for(int i=0; i < submatrix_dim; i++){

    tmp_index = block_clusterIndex[i];
    //__syncthreads();    


    tempBlockSum[tmp_index*numAttributes + threadIdx.x] += block_data[i*numAttributes + threadIdx.x];
    
    if(threadIdx.x==0){
      tempBlockclusterCount[tmp_index]++;
    }
    //__syncthreads();
  }
  
  //__syncthreads();
}


__global__ void calc_ClusterSizes(float *BlockClusterCount, int numCenters){

  float mean_sum = 0;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;

  if(tid<numCenters){

    mean_sum = BlockClusterCount[tid];
    __syncthreads();
#pragma unroll 8
    for(int i=1; i<NUMBER_OF_SUBMATRICES; i++){
      mean_sum += BlockClusterCount[i*numCenters + tid]; 
    }
    //__syncthreads();
    BlockClusterCount[tid] = mean_sum;
  }
}


__global__ void newCentroids(float *centroids, float *clusterSizes, int attributes, int numCenters){ 

  extern __shared__ float attrib[]; 
  
  attrib[threadIdx.x]  = centroids[blockIdx.x*attributes + threadIdx.x];  
  __syncthreads();
  
#pragma unroll 8
  for(int j=1; j<NUMBER_OF_SUBMATRICES; j++){
    attrib[threadIdx.x] += centroids[j*numCenters*attributes + blockIdx.x*attributes + threadIdx.x];
  }

  __syncthreads();

  centroids[blockIdx.x*attributes + threadIdx.x] = attrib[threadIdx.x] / clusterSizes[blockIdx.x]; 


}


void cluster(data_struct *data ,data_struct *clusters, data_struct *dataT){


  int iter = 0;
  int Cond, *d_Cond;
  int numObjects = data->leading_dim;
  int numAttributes = data->secondary_dim;
  int numClusters = clusters->secondary_dim;
  float *dataset = data->dataset;
  float *centroids = clusters->dataset;
  float *Index = data->members;
  float *clusterSizes = clusters->members;
  float *datasetT = dataT->dataset;  


#ifdef TIMEONLY
  float elapsedTime_kernel;
  hipEvent_t start_kernel, stop_kernel;
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
#endif


  hipMalloc((void**)&d_Cond, sizeof(int));
  
  float tmp_grid_size = ceil((float)numObjects/(float)BlockSize);
  float tmp_block_size = numObjects<BlockSize ? numObjects:BlockSize; 

  dim3 grid((int)tmp_grid_size,1);
  dim3 threads((int)tmp_block_size, 1);

  dim3 submatriceGrid(NUMBER_OF_SUBMATRICES, 1);  
  dim3 submatriceThreads(numAttributes, 1);

  tmp_grid_size = ceil((float)numClusters/(float)BlockSize);
  tmp_block_size = numClusters<BlockSize ? numClusters:BlockSize; 

  dim3 sizeGrid((int)tmp_grid_size, 1);
  dim3 sizeThreads((int)tmp_block_size, 1);

  dim3 newCentroidsGrid(numClusters, 1);
  dim3 newCentroidsThreads(numAttributes, 1);


#ifdef TIMEONLY
  hipEventRecord(start_kernel, 0);
#endif


  for(iter=0; iter<max_iterations; iter++){

    Cond = 0;
    hipMemcpy(d_Cond, &Cond, sizeof(int), hipMemcpyHostToDevice);

    CreateClusters<<<grid, threads, numAttributes*sizeof(float)>>>(dataset, centroids, Index, d_Cond, numObjects, numAttributes, numClusters);

    submaticesSum_kernel<<<submatriceGrid, submatriceThreads>>>(datasetT, Index, centroids, clusterSizes, numObjects, numAttributes, numClusters);

    calc_ClusterSizes<<<sizeGrid, sizeThreads>>>(clusterSizes, numClusters);

    newCentroids<<<newCentroidsGrid, newCentroidsThreads, numAttributes*sizeof(float)>>>(centroids, clusterSizes, numAttributes, numClusters);

    hipMemcpy(&Cond, d_Cond, sizeof(int), hipMemcpyDeviceToHost);

    if(Cond == 0){
      /*printf("\nCondition Reached, Process Terminating\n");*/
#ifndef TIMEONLY
      break;
#endif
    }

  }


#ifdef TIMEONLY
  hipEventRecord(stop_kernel, 0);  
  hipEventSynchronize(stop_kernel);
#endif


#ifdef TIMEONLY
  printf("\nFinised after %d iterations\n", iter);
#endif

#ifdef TIMEONLY
  hipEventElapsedTime(&elapsedTime_kernel, start_kernel, stop_kernel);
  printf("Time elapsed for kernel execution: %f ms\n", elapsedTime_kernel);
#endif

  hipFree(d_Cond);
#ifdef TIMEONLY
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel);
#endif
}

void mexFunction( int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){


  /*===== Host ======*/
  data_struct data_in;
  data_struct clusters;
  data_struct data_inT;
  //float SSE=0, new_SSE=0;
  double *numClusters_buf = 0;
  int numClusters = 0;
  float *execTime;

  //ptr2KernelFunction ptr2Func;

  /*==== Device ======*/
  data_struct d_data;
  data_struct d_clusters;
  data_struct d_dataT;

  /*===== Cuda Events===*/
  float elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  /*  
#ifdef TIMEONLY
  float elapsedTime_kernel;
  hipEvent_t start_kernel, stop_kernel;
  hipEventCreate(&start_kernel);
  hipEventCreate(&stop_kernel);
#endif
  */
  /*======== Initialization=======*/
  errorMessage1(nrhs);

  numClusters_buf = mxGetPr(prhs[2]);
  numClusters = (int)numClusters_buf[0];

  data_in.dataset = (float*)mxGetData(prhs[0]);
  data_in.leading_dim = mxGetM(prhs[0]);
  data_in.secondary_dim = mxGetN(prhs[0]);
  plhs[1] =  mxCreateNumericMatrix(data_in.leading_dim, 1, mxSINGLE_CLASS,mxREAL);
  data_in.members = (float*)mxGetData(plhs[1]);

  data_inT.dataset = (float*)mxGetData(prhs[1]);
  data_inT.leading_dim = mxGetM(prhs[1]);
  data_inT.secondary_dim = mxGetN(prhs[1]); 

  plhs[0] =  mxCreateNumericMatrix(data_in.secondary_dim, numClusters, mxSINGLE_CLASS, mxREAL);
  clusters.dataset = (float*)mxGetData(plhs[0]);
  clusters.leading_dim = data_in.secondary_dim;
  clusters.secondary_dim = numClusters;
  plhs[2] = mxCreateNumericMatrix(numClusters, 1, mxSINGLE_CLASS, mxREAL);
  clusters.members = (float*)mxGetData(plhs[2]);

  d_data.leading_dim = data_in.leading_dim;
  d_data.secondary_dim = data_in.secondary_dim;
  d_clusters.leading_dim = clusters.leading_dim;
  d_clusters.secondary_dim = clusters.secondary_dim;
  d_dataT.leading_dim = data_inT.leading_dim;
  d_dataT.secondary_dim = data_inT.secondary_dim;

  plhs[3] = mxCreateNumericMatrix(1, 1, mxSINGLE_CLASS, mxREAL);
  execTime = (float*)mxGetData(plhs[3]);

  errorMessage2(data_in.leading_dim, data_in.secondary_dim, numClusters);

  /*========= device memory allocation======*/

  hipMalloc((void**)&d_data.dataset, d_data.leading_dim*d_data.secondary_dim*sizeof(float));
  hipMalloc((void**)&d_data.members, d_data.leading_dim*sizeof(float));

  hipMalloc((void**)&d_clusters.dataset, d_clusters.leading_dim*d_clusters.secondary_dim*NUMBER_OF_SUBMATRICES*sizeof(float));
  hipMalloc((void**)&d_clusters.members, d_clusters.secondary_dim*NUMBER_OF_SUBMATRICES*sizeof(float));

  hipMalloc((void**)&d_dataT.dataset, d_dataT.leading_dim*d_dataT.secondary_dim*sizeof(float));
  hipMalloc((void**)&d_dataT.members, sizeof(float));


  /* initialize centroids*/
  if(nrhs == 4){
    float *centr_buff = (float*)mxGetData(prhs[3]);
    initialize_clusters(centr_buff, &clusters);
  }
  else{
    initialize_clusters_rand(&data_inT, &clusters);
  }

  hipEventRecord(start, 0);

  initialize_device_memory(&data_in, &d_data);
  initialize_device_memory(&clusters, &d_clusters);
  initialize_device_memory(&data_inT, &d_dataT);
  /*
#ifdef TIMEONLY
  hipEventRecord(start_kernel, 0);
#endif
  */
  cluster(&d_data, &d_clusters, &d_dataT);
  /*
#ifdef TIMEONLY
  hipEventRecord(stop_kernel, 0);  
  hipEventSynchronize(stop_kernel);
#endif
  */
  hipMemcpy(clusters.dataset, d_clusters.dataset, clusters.leading_dim*clusters.secondary_dim*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(data_in.members, d_data.members, data_in.leading_dim*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(clusters.members, d_clusters.members, clusters.secondary_dim*sizeof(float), hipMemcpyDeviceToHost);
  

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);


  hipEventElapsedTime(&elapsedTime, start, stop);

#ifdef TIMEONLY
  printf("Time elapsed: %f ms\n", elapsedTime);
#endif

  execTime[0] = elapsedTime; 
  /*
#ifdef TIMEONLY
  hipEventElapsedTime(&elapsedTime_kernel, start_kernel, stop_kernel);
  printf("Time elapsed for kernel execution: %f ms\n", elapsedTime_kernel);
#endif
  */
  /*==== clean device===*/
  cleanDevice(&d_data);
  cleanDevice(&d_clusters);
  cleanDevice(&d_dataT);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  /*
#ifdef TIMEONLY
  hipEventDestroy(start_kernel);
  hipEventDestroy(stop_kernel);
#endif
  */
}

